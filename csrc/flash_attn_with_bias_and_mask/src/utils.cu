#include "hip/hip_runtime.h"
#include "utils.h"

void SetZero(void *ptr, size_t sizeof_type, std::initializer_list<int> shapes, hipStream_t stream) {
    size_t n = sizeof_type;
    for (int s : shapes) n *= s;
    FMHA_CHECK_CUDA(hipMemsetAsync(ptr, 0, n, stream));
}

template <typename T>
static __global__ void FillConstantKernel(T *ptr, T value, size_t n) {
  auto idx = static_cast<size_t>(blockDim.x) * blockIdx.x + threadIdx.x;
  if (idx < n) {
    ptr[idx] = value;
  }
} 

template <typename T>
void SetConstValue(void *ptr, T value, size_t n, hipStream_t stream) {
  constexpr auto kNumThreads = 1024;
  auto block = (n + kNumThreads - 1) / kNumThreads; 
  FillConstantKernel<T><<<block, kNumThreads, 0, stream>>>(static_cast<T *>(ptr), value, n);
} 

template
void SetConstValue(void *ptr, float value, size_t n, hipStream_t stream);

static __global__ void _float2half(float *float_ptr, __half *half_ptr, size_t n) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < n) {
    half_ptr[idx] = __float2half(float_ptr[idx]);
  }
}

void Float2Half(void *float_ptr, void *half_ptr, size_t n, hipStream_t stream) {
  constexpr auto kNumThreads = 1024;
  auto block = (n + kNumThreads - 1) / kNumThreads; 
  _float2half<<<block, kNumThreads, 0, stream>>>(static_cast<float *>(float_ptr), static_cast<__half *>(half_ptr), n);
} 

static __global__ void _float2bfloat16(float *float_ptr, __hip_bfloat16 *bf16_ptr, size_t n) {
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if (idx < n) {
    bf16_ptr[idx] = __float2bfloat16(float_ptr[idx]);
  }
}

void Float2BF16(void *float_ptr, void *bf16_ptr, size_t n, hipStream_t stream) {
  constexpr auto kNumThreads = 1024;
  auto block = (n + kNumThreads - 1) / kNumThreads; 
  _float2bfloat16<<<block, kNumThreads, 0, stream>>>(static_cast<float *>(float_ptr), static_cast<__hip_bfloat16 *>(bf16_ptr), n);
} 
