// Copyright (c) 2022, Tri Dao.

#include "fmha_bwd_with_mask_bias_launch_template.h"

bool run_fmha_bwd_with_mask_bias_hdim64(FMHA_dgrad_params &params, hipStream_t stream) {
    bool status = true;
    auto dprops = GetDeviceProperties(-1);
    FP16_SWITCH(params.is_bf16, ([&] {
        if( params.seqlen_k == 128 ) {
            using Kernel_traits = FMHA_kernel_traits<128, 64, 16, 1, 8, 0x08u, elem_type>;
            status = run_fmha_dgrad_fp16_sm80_loop_<Kernel_traits>(params, stream);
        } else if( params.seqlen_k >= 256 ) {
            if (dprops->major == 8 && dprops->minor == 0) {
                // Don't share smem for K & V, and don't keep V in registers
                // This speeds things up by 2-3% by avoiding register spills, but it
                // uses more shared memory, which is fine on A100 but not other GPUs.
                // For other GPUs, we keep V in registers.
                using Kernel_traits = FMHA_kernel_traits<256, 64, 16, 1, 8, 0x100u, elem_type>;
                status = run_fmha_dgrad_fp16_sm80_loop_<Kernel_traits>(params, stream);
            } else if (dprops->major == 8 && dprops->minor > 0) {
                using Kernel_traits = FMHA_kernel_traits<256, 64, 16, 1, 8, 0x08u, elem_type>;
                status = run_fmha_dgrad_fp16_sm80_loop_<Kernel_traits>(params, stream);
            } else if (dprops->major == 7 && dprops->minor == 5) {
                using Kernel_traits = FMHA_kernel_traits<128, 64, 16, 1, 8, 0x08u, elem_type>;
                status = run_fmha_dgrad_fp16_sm80_loop_<Kernel_traits>(params, stream);
            }
        }
    }));
    return status;
}
